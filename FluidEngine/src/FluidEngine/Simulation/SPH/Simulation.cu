#include "Simulation.cuh"
#include <hip/hip_runtime.h>

namespace fe {
	extern "C" {
		void SetParameters(SimParams* params) {
			hipMemcpyToSymbol(HIP_SYMBOL(simulationParameters), params, sizeof(SimParams));
		}
	}
}