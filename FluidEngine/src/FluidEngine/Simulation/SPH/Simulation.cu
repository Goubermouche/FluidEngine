#include "hip/hip_runtime.h"
#include "Simulation.cuh"
#include <hip/hip_runtime.h>
#include <FluidEngine/Compute/Utility/CUDAGLInterop.h>
#include <iostream>
#include <FluidEngine/Compute/Utility/cutil.h>

namespace fe {
	extern "C" {
		int IDivUp(int a, int b) {
			return a % b != 0 ? a / b + 1 : a / b;
		}

		void ComputeGridSize(int n, int blockSize, int& blockCount, int& threadCount) {
			threadCount = min(blockSize, n);
			blockCount = IDivUp(n, threadCount);
		}

		void Integrate(float4* newPos, float4* newVel, float4* oldPos, float4* oldVel, int particleCount)
		{
			int numThreads;
			int numBlocks;
			ComputeGridSize(particleCount, 256, numBlocks, numThreads);

			IntegrateKernel <<< numBlocks, numThreads >>> (newPos, newVel, oldPos, oldVel);
			CUT_CHECK_ERROR("Kernel execution failed: IntegrateKernel");
		}

		void Hash(float4* pos, uint2* particleHash, int particleCount) 
		{
			int threadCount;
			int blockCount;
			ComputeGridSize(particleCount, 256, blockCount, threadCount);

			CalculateHashKernel <<< blockCount, threadCount >>> (pos, particleHash);
			CUT_CHECK_ERROR("Kernel execution failed: CalculateHashKernel");
		}

		void Reorder(uint2* particleHash, uint* cellStart, float4* oldPos, float4* oldVel, float4* sortedPos, float4* sortedVel, int particleCount, int cellCount)
		{
			int threadCount;
			int blockCount;
			ComputeGridSize(particleCount, 256, blockCount, threadCount);
			CUDA_SAFE_CALL(hipMemset(cellStart, 0xffffffff, cellCount * sizeof(uint)));
			ReorderKernel <<< blockCount, threadCount >>> (particleHash, cellStart, oldPos, oldVel, sortedPos, sortedVel);
			CUT_CHECK_ERROR("Kernel execution failed: ReorderKernel");
		}
	}
}