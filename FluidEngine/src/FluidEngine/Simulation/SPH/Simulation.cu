#include "FluidEngine/Compute/Utility/CUDA/cutil.h"
#include "FluidEngine/Compute/Utility/RadixSort/RadixSort.cuh"
#include "FluidEngine/Simulation/SPH/SimulationKernel.cu"
#include "FluidEngine/Compute/Utility/CudaKernelUtility.cuh"

#include <glad/glad.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

namespace fe {
	extern "C" {
		void SetParameters(SimulationData& params) {
			printf("parameters set!\n");
			CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_Description), &params, sizeof(SimulationData)));
		}

		void Integrate(unsigned int oldPositionVBO, unsigned int newPositionVBO, glm::vec4* oldVelocity, glm::vec4* newVelocity, int particleCount)
		{
			// Grid size
			int threadCount;
			int blockCount;
			ComputeGridSize(particleCount, 256, blockCount, threadCount);

			// Buffer data
			glm::vec4* oldPosition;
			glm::vec4* newPosition;
			CUDA_SAFE_CALL(cudaGLMapBufferObject((void**)&oldPosition, oldPositionVBO));
			CUDA_SAFE_CALL(cudaGLMapBufferObject((void**)&newPosition, newPositionVBO));

			// Kernel
			IntegrateKernel <<< blockCount, threadCount >>> (newPosition, oldPosition, newVelocity, oldVelocity);
			CUT_CHECK_ERROR("Kernel execution failed: IntegrateKernel");

			// Unbind buffers
			CUDA_SAFE_CALL(cudaGLUnmapBufferObject(oldPositionVBO));
			CUDA_SAFE_CALL(cudaGLUnmapBufferObject(newPositionVBO));

			CUDA_SAFE_CALL(hipDeviceSynchronize())
		}

		void CalculateHash(unsigned int positionVBO, glm::uvec2* particleHash, int particleCount)
		{
			// Grid size
			int threadCount;
			int blockCount;
			ComputeGridSize(particleCount, 512, blockCount, threadCount);

			// Buffer data
			glm::vec4* position;
			CUDA_SAFE_CALL(cudaGLMapBufferObject((void**)&position, positionVBO));

			// Kernel
			CalculateHashKernel <<< blockCount, threadCount >>> (position, particleHash);
			CUT_CHECK_ERROR("Kernel execution failed: CalculateHashKernel");

			// Unbind buffers
			CUDA_SAFE_CALL(cudaGLUnmapBufferObject(positionVBO));

			CUDA_SAFE_CALL(hipDeviceSynchronize());
		}

		void Reorder(unsigned int oldPositionVBO, glm::vec4* oldVelocity, glm::vec4* sortedPosition, glm::vec4* sortedVelocity,
			glm::uvec2* particleHash, unsigned int* cellStart, unsigned int particleCount, unsigned int cellCount)
		{
			// Grid size
			int threadCount;
			int blockCount;
			ComputeGridSize(particleCount, 256, blockCount, threadCount);

			// Set all indices of the array to '0xffffffff'
			CUDA_SAFE_CALL(hipMemset(cellStart, 0xffffffff, cellCount * sizeof(unsigned int)));

			// Buffer data
			glm::vec4* oldPosition;
			CUDA_SAFE_CALL(cudaGLMapBufferObject((void**)&oldPosition, oldPositionVBO));

			// Texture data
			unsigned int float4MemorySize = particleCount * sizeof(glm::vec4);
			CUDA_SAFE_CALL(hipBindTexture(0, oldPositionTexture, oldPosition, float4MemorySize));
			CUDA_SAFE_CALL(hipBindTexture(0, oldVelocityTexture, oldVelocity, float4MemorySize));

			// Kernel
			ReorderKernel << < blockCount, threadCount >> > (particleHash, cellStart, oldPosition, oldVelocity, sortedPosition, sortedVelocity);
			CUT_CHECK_ERROR("Kernel execution failed: ReorderKernel");

			// Unbind textures
			CUDA_SAFE_CALL(hipUnbindTexture(oldPositionTexture));
			CUDA_SAFE_CALL(hipUnbindTexture(oldVelocityTexture));

			// Unbind buffers
			CUDA_SAFE_CALL(cudaGLUnmapBufferObject(oldPositionVBO));

			CUDA_SAFE_CALL(hipDeviceSynchronize());
		}

		void Collide(unsigned int positionVBO, glm::vec4* sortedPosition, glm::vec4* sortedVelocity,
			glm::vec4* oldVelocity, glm::vec4* newVelocity, float* pressure, float* density,
			glm::uvec2* particleHash, unsigned int* cellStart, unsigned int particleCount, unsigned int cellCount)
		{
			// Grid size
			int threadCount;
			int blockCount;
			ComputeGridSize(particleCount, 64, blockCount, threadCount);

			// Buffer data
			glm::vec4* newPosition;
			CUDA_SAFE_CALL(cudaGLMapBufferObject((void**)&newPosition, positionVBO));

			// Texture data
			unsigned int float4MemorySize = particleCount * sizeof(glm::vec4);
			unsigned int float1MemorySize = particleCount * sizeof(float);
			CUDA_SAFE_CALL(hipBindTexture(0, oldPositionTexture, sortedPosition, float4MemorySize));
			CUDA_SAFE_CALL(hipBindTexture(0, oldVelocityTexture, sortedVelocity, float4MemorySize));
			CUDA_SAFE_CALL(hipBindTexture(0, pressureTexture, pressure, float1MemorySize));
			CUDA_SAFE_CALL(hipBindTexture(0, densityTexture, density, float1MemorySize));
			CUDA_SAFE_CALL(hipBindTexture(0, particleHashTexture, particleHash, particleCount * sizeof(glm::uvec2)));
			CUDA_SAFE_CALL(hipBindTexture(0, cellStartTexture, cellStart, cellCount * sizeof(unsigned int)));

			// Kernel
			CalculateDensityKernel <<< blockCount, threadCount >>> (sortedPosition, pressure, density, particleHash, cellStart);
			CUT_CHECK_ERROR("Kernel execution failed: CalculateDensityKernel");

			CUDA_SAFE_CALL(hipDeviceSynchronize());

			// Kernel
			CalculateForceKernel <<< blockCount, threadCount >>> (newPosition, newVelocity, sortedPosition, sortedVelocity, pressure, density, particleHash, cellStart);
			CUT_CHECK_ERROR("Kernel execution failed: CalculateForceKernel");

			// Unbind buffers
			CUDA_SAFE_CALL(cudaGLUnmapBufferObject(positionVBO));

			// Unbind textures
			CUDA_SAFE_CALL(hipUnbindTexture(oldPositionTexture));
			CUDA_SAFE_CALL(hipUnbindTexture(oldVelocityTexture));
			CUDA_SAFE_CALL(hipUnbindTexture(pressureTexture));
			CUDA_SAFE_CALL(hipUnbindTexture(densityTexture));
			CUDA_SAFE_CALL(hipUnbindTexture(particleHashTexture));
			CUDA_SAFE_CALL(hipUnbindTexture(cellStartTexture));

			CUDA_SAFE_CALL(hipDeviceSynchronize());
		}
	}
}