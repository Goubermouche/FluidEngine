#include "SimulationKernel.cuh"
#include "cutil/inc/cutil.h"
#include <iostream>

namespace fe {
	void SetParameters(SimulationParameters& params)
	{
		printf("parameters set!\n");
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(parameters), &params, sizeof(SimulationParameters)));
	}
}